#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#define NUM_THREADS 256

typedef thrust::complex<double> Complex;

Complex *srft_c, *w_c, *kw_c, *dft_c, *dct_c, *fwht_c, *a_c, *dct_shift_c, *b_c, *d_c;
double *fwht_r, *a_gpu, *sa_re_gpu, *sa_im_gpu, *d_r, *srft_r;
int *bit_cnt, *bit_rev, *kbit_rev, k;
int *f_gpu, *perm_gpu, *r_gpu;

bool flag;

__global__ void transpose(const double *a, double *temp, int N, int k, int m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) temp[i] = a[(i % k) * m + (i / k)];
}

__global__ void transpose(const Complex *a, Complex *temp, int N, int k, int m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) temp[i] = a[(i % k) * m + (i / k)];
}

__global__ void compute_w(Complex* w_c, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i <= N) w_c[i] = Complex(cos(2 * M_PI * i / N), sin(2 * M_PI * i / N));
}

__global__ void compute_bit_rev(int* bit_rev, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    int j = 0;
    for (int x = (N >> 1), y = i; x; x >>= 1, y >>= 1) j = ((j << 1) | (y & 1));
    bit_rev[i] = j;
}

__global__ void compute_bitcount(int *bit_cnt, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    int temp = (i & 0x55555555) + ((i >> 1) & 0x55555555);
    temp = (temp & 0x33333333) + ((temp >> 2) & 0x33333333);
    temp = (temp & 0x0F0F0F0F) + ((temp >> 4) & 0x0F0F0F0F);
    temp = (temp & 0x00FF00FF) + ((temp >> 8) & 0x00FF00FF);
    temp = (temp & 0x0000FFFF) + ((temp >> 16) & 0x0000FFFF);
    bit_cnt[i] = temp;
}

__global__ void compute_dct_shift(Complex *dct_shift_c, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    dct_shift_c[i] = Complex(2 * cos(PI * i / 2. / N), 2 * sin(PI * i / 2. / N));
}

__global__ void fwht_butterfly(double *a_gpu, int N, int h) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N / 2) return;
    int p = (i / h) * h * 2, q = i % h;
    double x = a_gpu[p + q], y = a_gpu[p + q + h];
    a_gpu[p + q] = x + y;
    a_gpu[p + q + h] = x - y;
}

__device__ void fwht(double* a, int N) {
    for (int h = 1; h < N; h *= 2) {
        for (int i = 0; i < N; i += h * 2) {
            for (int j = 0; j < h; ++j) {
                double x = a[i + j];
                double y = a[i + j + h];
                a[i + j] = x + y;
                a[i + j + h] = x - y;
            }
        }
    }
}

void fwht_parallel(double* a, int N) {
    for (int h = 1; h < N; h *= 2) {
        fwht_butterfly<<<(N / 2 + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(a, N, h);
    }
}

__global__ void fwht_block(double *a, int N, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i * k >= N) return;
    fwht(a + i * k, k);
}

__global__ void fwht_nlogd_compute(double *d_r, double *fwht_r, int d, int m, const int *r, int k, const int *bit_cnt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d * m) return;
    int it = i / m, p = i % m;
    int j = r[it], y = j / m;
    double v = fwht_r[p * k + y];
    double x = (bit_cnt[j & p] ? -v : v);
    d_r[i] = x;
}

__global__ void fwht_nlogd_store(double *res_r, double *d_r, int d, int m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d) return;
    res_r[i] = d_r[i + m] - d_r[i];
}

void fwht_nlogd(double* a, int N, int k, int d, const int *r) {
    int m = N / k;
    transpose<<<(N + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(a, fwht_r, N, k, m);
    if (flag) {
        for (int i = 0; i < N; i += k) fwht_parallel(fwht_r + i, k);
    } else {
        fwht_block<<<(N / k + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(fwht_r, N, k);
    }
    fwht_nlogd_compute<<<(d * m + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(d_r, fwht_r, d, m, r, k, bit_cnt);
    thrust::inclusive_scan(d_r, d_r + d * m, d_r);
    fwht_nlogd_store<<<(d + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(a, d_r, d, m);
}

__device__ void fft(Complex *a_c, int N, const Complex *w_c, const int *bit_rev) {
    for (int i = 0; i < N; ++i) {
        int j = bit_rev[i];
        if (i < j) std::swap(a_c[i], a_c[j]);
    }
    for (int m = 2; m <= N; m *= 2) {
        int gap = m / 2, step = N / m;
        for (int i = 0; i < N; i += m) {
            const Complex *o = w_c;
            for (int j = i; j < i + gap; ++j, o += step) {
                Complex u = a_c[j], v = *o * a_c[j + gap];
                a_c[j] = u + v;
                a_c[j + gap] = u - v;
            }
        }
    }
}

__global__ void fft_bit_rev(const Complex *a_gpu, Complex *b_gpu, int N, const int *bit_rev) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = bit_rev[i];
    b_gpu[j] = a_gpu[i];
}

__global__ void fft_butterfly(Complex *a_c, int gap, int step, const Complex *w_c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int p = (i / gap) * gap * 2, q = i % gap;
    int x = p + q, y = p + q + gap;
    Complex u = a_c[x], v = w_c[q * step] * a_c[y];
    a_c[x] = u + v;
    a_c[y] = u - v;
}

__global__ void fft_block(Complex *a_c, int N, const Complex *w_c, const int *bit_rev, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i * k >= N) return;
    fft(a_c + i * k, k, w_c, bit_rev);
}

void fft_parallel(Complex *a_c, int N, const Complex *w_c, const int *bit_rev) {
    if (N <= 2 * NUM_THREADS) {
        fft_block<<<1, 1>>>(a_c, N, w_c, bit_rev, N);
        return;
    }
    fft_bit_rev<<<(N + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(a_c, b_c, N, bit_rev);
    hipMemcpy(a_c, b_c, N * sizeof(Complex), hipMemcpyDeviceToDevice);
    for (int m = 2; m <= N; m *= 2) {
        int gap = m / 2, step = N / m;
        fft_butterfly<<<(N / NUM_THREADS) / 2, NUM_THREADS>>>(a_c, gap, step, w_c);
    }
}

__global__ void dft_nlogd_compute(Complex *res_c, Complex *dft_c, Complex *w_c, int d, int m, const int *r, int N, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d * m) return;
    int it = i / m, p = i % m;
    int j = r[it];
    int x = ((int64_t)j * p) % N, y = j % k;
    Complex v = dft_c[p * k + y];
    res_c[i] = v * w_c[x];
}

__global__ void dft_nlogd_store(Complex *res_c, Complex *d_c, int d, int m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d) return;
    res_c[i] = d_c[i + m] - d_c[i];
}

void dft_nlogd(Complex* a_c, int N, int k, int d, const int *r) {
    int m = N / k;
    transpose<<<(N + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(a_c, dft_c, N, k, m);
    if (flag) {
        for (int i = 0; i < N; i += k) fft_parallel(dft_c + i, k, kw_c, kbit_rev);
    } else {
        fft_block<<<(N / k + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(dft_c, N, kw_c, kbit_rev, k);
    }
    dft_nlogd_compute<<<(d * m + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(d_c, dft_c, w_c, d, m, r, N, k);
    thrust::inclusive_scan(d_c, d_c + d * m, d_c);
    dft_nlogd_store<<<(d + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(a_c, d_c, d, m);
}

void dft(Complex *srft_c, int N) {
    fft_parallel(srft_c, N, w_c, bit_rev);
}

__global__ void dct_store(Complex *dct_c, Complex *a_c, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    dct_c[(i & 1) ? N - 1 - (i >> 1) : (i >> 1)] = a_c[i];
}

__global__ void dct_load(Complex *a_c, Complex *dct_c, Complex *dct_shift_c, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    a_c[i] = (dct_c[i] * dct_shift_c[i]).real();
}

void dct(Complex *a_c, int N) {
    dct_store<<<(N + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(dct_c, a_c, N);
    fft_parallel(dct_c, N, w_c, bit_rev);
    dct_load<<<(N + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(a_c, dct_c, dct_shift_c, N);
}

__global__ void dct_nlogd_load(Complex *a_c, Complex *dct_c, Complex *dct_shift_c, int d, const int *r) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d) return;
    int j = r[i];
    a_c[i] = (dct_c[i] * dct_shift_c[j]).real();
}

void dct_nlogd(Complex *a, int N, int k, int d, const int *r) {
    dct_store<<<(N + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(dct_c, a_c, N);
    dft_nlogd(dct_c, N, k, d, r);
    dct_nlogd_load<<<(d + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(a_c, dct_c, dct_shift_c, d, r);
}

/*
* @params N: array size
* @params d: r size
* @params n_ranks: num_ranks
* @params f: vector of N random signs (-1 or +1)
* @params perm: random permutation of [0, N)
* @params a: array to be srft'd
* @params r: d random elements from [0, N) (to be rd)
* @params sa_re: destination to store real part of result
* @params sa_im: destination to store imaginary part of result
* @params transform: the transformation to be performed
*/

void init(int N, int d, int n_ranks, const int *f, const int *perm, const int *r, Transform transform) {
    hipMalloc((void**) &srft_c, N * sizeof(Complex));
    hipMalloc((void**) &f_gpu, N * sizeof(int));
    hipMalloc((void**) &perm_gpu, N * sizeof(int));
    hipMalloc((void**) &r_gpu, d * sizeof(int));
    hipMemcpy(f_gpu, f, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(perm_gpu, perm, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(r_gpu, r, N * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**) &a_gpu, N * sizeof(double));
    hipMalloc((void**) &sa_re_gpu, d * sizeof(double));
    hipMalloc((void**) &sa_im_gpu, d * sizeof(double));
    if (transform == Transform::walsh) {
        hipMalloc((void**) &srft_r, N * sizeof(double));
    }
    if (transform == Transform::fourier || transform == Transform::cosine) {
        hipMalloc((void**) &b_c, N * sizeof(Complex));
        hipMalloc((void**) &dft_c, N * sizeof(Complex));
        hipMalloc((void**) &w_c, (N + 1) * sizeof(Complex));
        hipMalloc((void**) &bit_rev, N * sizeof(int));
        compute_w<<<(N + NUM_THREADS) / NUM_THREADS, NUM_THREADS>>>(w_c, N);
        hipMemcpy(w_c + N, w_c, sizeof(Complex), hipMemcpyDeviceToDevice);
        compute_bit_rev<<<(N + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(bit_rev, N);
    }
    if (transform == Transform::cosine) {
        hipMalloc((void**) &dct_c, N * sizeof(Complex));
        hipMalloc((void**) &dct_shift_c, N * sizeof(Complex));

        compute_dct_shift<<<(N + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(dct_shift_c, N);
    }
}

void init_nlogd(int N, int d, int n_ranks, const int *f, const int *perm, const int *r, Transform transform) {
    k = 2;
    for (int i = 1; k < d * i && k < N; ++i) k *= 2;
    init(N, d, n_ranks, f, perm, r, transform);
    if (transform == Transform::fourier || transform == Transform::cosine) {
        hipMalloc((void**) &kw_c, (k + 1) * sizeof(Complex));
        hipMalloc((void**) &kbit_rev, k * sizeof(int));
        hipMalloc((void**) &d_c, N * sizeof(Complex));
        compute_w<<<(k + NUM_THREADS) / NUM_THREADS, NUM_THREADS>>>(kw_c, k);
        compute_bit_rev<<<(k + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(kbit_rev, k);
    }
    if (transform == Transform::walsh) {
        hipMalloc((void**) &bit_cnt, N * sizeof(int));
        hipMalloc((void**) &fwht_r, N * sizeof(double));
        hipMalloc((void**) &d_r, N * sizeof(double));
        compute_bitcount<<<(N + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(bit_cnt, N);
    }
    //flag = (N < (double)k * k * (31 - __builtin_clz(k)));
    flag = false;
}

__global__ void shuffle(Complex *srft_c, double *a_gpu, int *perm_gpu, int *f_gpu, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    srft_c[i] = Complex(a_gpu[perm_gpu[i]] * f_gpu[i], 0);
}

__global__ void shuffle_real(double *srft_r, double *a_gpu, int *perm_gpu, int *f_gpu, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    srft_r[i] = a_gpu[perm_gpu[i]] * f_gpu[i];
}

__global__ void srft_save(double *sa_re_gpu, double *sa_im_gpu, double scale, Complex *srft_c, int d, const int *r) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d) return;
    sa_re_gpu[i] = srft_c[r[i]].real();
    sa_im_gpu[i] = srft_c[r[i]].imag();
}

__global__ void srft_save(double *sa_re_gpu, double *sa_im_gpu, double scale, Complex *srft_c, int d) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d) return;
    sa_re_gpu[i] = srft_c[i].real();
    sa_im_gpu[i] = srft_c[i].imag();
}

__global__ void srft_real_save(double *sa_re_gpu, double scale, double *srft_r, int d, const int *r) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= d) return;
    sa_re_gpu[i] = srft_r[r[i]];
}

void srft(int N, int d, int n_ranks, const int *f, const int *perm, const double *a, double *sa_re, double *sa_im, const int *r, Transform transform) {
    hipMemcpy(a_gpu, a, N * sizeof(double), hipMemcpyHostToDevice);
    if (transform == Transform::walsh) {
        shuffle_real<<<(N + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(srft_r, a_gpu, perm_gpu, f_gpu, N);
        fwht_parallel(srft_r, N);
        hipMemcpy(sa_re, srft_r, d * sizeof(double), hipMemcpyDeviceToHost);
    } else {
        shuffle<<<(N + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(srft_c, a_gpu, perm_gpu, f_gpu, N);
        if (transform == Transform::fourier) {
            dft(srft_c, N);
        } else {
            assert(transform == Transform::cosine);
            dct(srft_c, N);
        }
        double scale = sqrt((double)N / d);
        srft_save<<<(d + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(sa_re_gpu, sa_im_gpu, scale, srft_c, d, r);
        hipMemcpy(sa_re, sa_re_gpu, d * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(sa_im, sa_im_gpu, d * sizeof(double), hipMemcpyDeviceToHost);
    }
}

void srft_nlogd(int N, int d, int n_ranks, const int *f, const int *perm, const double *a, double *sa_re, double *sa_im, const int *r, Transform transform) {
    hipMemcpy(a_gpu, a, N * sizeof(double), hipMemcpyHostToDevice);
    if (transform == Transform::walsh) {
        shuffle_real<<<(N + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(srft_r, a_gpu, perm_gpu, f_gpu, N);
        fwht_nlogd(srft_r, N);
        hipMemcpy(sa_re, srft_r, d * sizeof(double), hipMemcpyDeviceToHost);
    } else {
        shuffle<<<(N + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(srft_c, a_gpu, perm_gpu, f_gpu, N);
        if (transform == Transform::fourier) {
            dft(srft_c, N);
        } else {
            assert(transform == Transform::cosine);
            dct(srft_c, N);
        }
        double scale = sqrt((double)N / d);
        srft_save<<<(d + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(sa_re_gpu, sa_im_gpu, scale, srft_c, d);
        hipMemcpy(sa_re, sa_re_gpu, d * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(sa_im, sa_im_gpu, d * sizeof(double), hipMemcpyDeviceToHost);
    }
}
